
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>




typedef unsigned long long data_t;

static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)



template <class T>
void cuda_memcpy(T* target, const T* source, std::size_t num, hipMemcpyKind direction) {
CHECK(hipMemcpy(target, source, num * sizeof(T), direction));
}

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
  }


__global__ void getMask(data_t *d_in, unsigned int *d_out, const int len, const unsigned int n, unsigned int bit_shift, unsigned int One) {
    unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int bit = 0;
    unsigned int start=index*len;
    if (start>=n) return;
    unsigned int end=start+len;
    for(unsigned int i=start;i<end && i<n; i++ ){
      bit=d_in[i]&(1 << bit_shift);
      bit = (bit > 0) ? 1 : 0;
      d_out[i] = (One ? bit : 1 - bit);
    }
}

__global__ void getIndex(unsigned int *d_index, unsigned int *d_sum, unsigned int *d_mask, const int len, const unsigned int n,
    unsigned int total_pre) {
    unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
    
    unsigned int start=index*len;
  
    if (start>=n || total_pre==n) return;
    
    if (start==0) {
      d_index[start]=total_pre;
      start++;
    }
    unsigned int end=start+len;
    for (unsigned int i=start; i<end && i<n; i++){
      if(d_mask[i]==1){
        d_index[i]=total_pre+d_sum[i-1];
      }
    }
    // if (index < n) {
    //     if (d_mask[index] == 1) {
    //         d_index[index] = total_pre + d_sum[index];
    //     }
    // }
}
// scatter<<<divup(n,block_size*len),block_size>>>(d_in, d_index, d_out, len, n);
__global__ void scatter(data_t *d_in, unsigned int *d_index, data_t *d_out, const int len, const unsigned int n) {
    unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;

    unsigned int start=index*len;
    if (start>=n) return;
    unsigned int end=start+len;

    for(unsigned int i=start;i<end && i<n; i++ ){
      d_out[d_index[i]]=d_in[i];
    }
    // if (index < n) {
    //     d_out[d_index[index]] = d_in[index];
    // }
}


__global__ void prefixsum(unsigned int* mask, unsigned int* output,const int len, const unsigned int n ){
  unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
  int step=len;
  int start=index*len;
  if (start>=n) return;
  int end=start+step;
  output[start]=mask[start];
  for(unsigned int i=start+1;i<end&&i<n;i++){
    output[i]+=output[i-1]+mask[i];
  }
}


__global__ void serialsum_accrossthread(unsigned int* sum,const int len, const unsigned int n){
  unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
  int step=len;
  int offset=2*step-1;
  unsigned int start=step*blockDim.x*index+offset;
  unsigned int end=step*blockDim.x*(index+1);
  for(unsigned int i=start;i<end && i<n; i+=step){
    sum[i]+=sum[i-step];
  }
}

__global__ void mergethread(unsigned int* sum,const int len, const unsigned int n){
  if (threadIdx.x==0) return;

  unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
  int step=len;
  unsigned int start=index*step;
  unsigned int end=start+step-1;
  unsigned int base=sum[start-1];

  for(unsigned int i=start; i<end && i<n; i++){
    sum[i]+=base;
  }

}

__global__ void serialsum_accrossblock(unsigned int* sum,const int len, const unsigned int n){
  

  unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
  int step=len*blockDim.x;
  int offset=2*step-1;
  unsigned int start= blockDim.x*step*index+offset;
  unsigned int end= blockDim.x*step*(index+1);
  for(unsigned int i=start; i<end && i<n; i+=step){
    sum[i]+=sum[i-step];
  }
}


__global__ void mergeblock(unsigned int* sum,const int len, const unsigned int n){
  if (blockIdx.x==0) return;

  unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
  int step=len;
  int start=index*step;
  int end=start+step;
  int base=sum[blockIdx.x*len*blockDim.x-1];//last element at last block
  for(int i=start; i<end && i<n; i++){
    sum[i]+=base;
  }
}

void psort(int n, data_t *data) {
  // FIXME: Implement a more efficient parallel sorting algorithm for the GPU.

  const int block_size=64;//64 threads per block;
  const int len=1000; // add 1000 prefix sum per thread; 

  data_t *d_temp;
  data_t *d_in=NULL;
  CHECK(hipMalloc((void**)&d_in,n*sizeof(data_t)));

  data_t *d_out_long=NULL;
  CHECK(hipMalloc((void**)&d_out_long,n*sizeof(data_t)));
  unsigned int *d_out=NULL;
  CHECK(hipMalloc((void**)&d_out,n*sizeof(unsigned int)));
  unsigned int *d_sum=NULL;
  CHECK(hipMalloc((void**)&d_sum,n*sizeof(unsigned int)));
  unsigned int *d_index=NULL;
  CHECK(hipMalloc((void**)&d_index,n*sizeof(unsigned int)));

  data_t cal_result[n];
  unsigned int index[n];
  cuda_memcpy(d_in,data,n,hipMemcpyHostToDevice);

  int bits=sizeof(data_t);



  unsigned int total_zeros;

  
  for(int i=0; i<bits; i++){
      // get mask for 0 and store in d_out
      // getMask<<<dimGrid, dimBlock>>>(d_in, d_out, n, i, 0);
      CHECK(hipMemset(d_sum,0,n*sizeof(unsigned int)));
      getMask<<<divup(n,block_size*len),block_size>>>(d_in, d_out, len, n, i, 0);
      std::cout<<"out"<<std::endl;
      CHECK(hipMemcpy(index,d_out, n * sizeof(unsigned int), hipMemcpyDeviceToHost));
      for(int j=0; j<n; j++){
        std::cout<< index[j] << " " ;
      }
      std::cout<< std::endl;

      CHECK(hipGetLastError());
      //inclusive prefix sum
      
      prefixsum<<<divup(n,block_size*len),block_size>>>(d_out,d_sum,len,n);
      CHECK(hipGetLastError());
      serialsum_accrossthread<<<divup(n,block_size*len*block_size),block_size>>>(d_sum,len,n);
      CHECK(hipGetLastError());
      mergethread<<<divup(n,block_size*len),block_size>>>(d_sum,len,n);
      CHECK(hipGetLastError());
      serialsum_accrossblock<<<divup(n,block_size*len*block_size*block_size) ,block_size>>>(d_sum,len,n);
      CHECK(hipGetLastError());
      mergeblock<<<divup(n,block_size*len),block_size>>>(d_sum,len,n);
      CHECK(hipGetLastError());
      CHECK(hipMemcpy(&total_zeros, d_sum+n-1, sizeof(unsigned int), hipMemcpyDeviceToHost));
      std::cout<< "zeros" << total_zeros<< std::endl;
      std::cout<<"sum1"<<std::endl;
      CHECK(hipMemcpy(index,d_sum, n * sizeof(unsigned int), hipMemcpyDeviceToHost));
      for(int j=0; j<n; j++){
        std::cout<< index[j] << " " ;
      }
      std::cout<< std::endl;

      getIndex<<<divup(n,block_size*len),block_size>>>(d_index, d_sum, d_out, len, n, 0);
      
      CHECK(hipGetLastError());
      // get mask for 1 and store in d_out
      getMask<<<divup(n,block_size*len),block_size>>>(d_in, d_out, len, n, i, 1);

      // std::cout<<"out"<<std::endl;
      // CHECK(cudaMemcpy(index,d_out, n * sizeof(unsigned int), cudaMemcpyDeviceToHost));
      // for(int j=0; j<n; j++){
      //   std::cout<< index[j] << " " ;
      // }
      // std::cout<< std::endl;

      CHECK(hipGetLastError());
      //inclusive prefix sum
      CHECK(hipMemset(d_sum,0,n*sizeof(unsigned int)));
      prefixsum<<<divup(n,block_size*len),block_size>>>(d_out,d_sum,len,n);
      CHECK(hipGetLastError());
      serialsum_accrossthread<<<divup(n,block_size*len*block_size),block_size>>>(d_sum,len,n);
      CHECK(hipGetLastError());
      mergethread<<<divup(n,block_size*len),block_size>>>(d_sum,len,n);
      CHECK(hipGetLastError());
      serialsum_accrossblock<<<divup(n,block_size*len*block_size*block_size) ,block_size>>>(d_sum,len,n);
      CHECK(hipGetLastError());
      mergeblock<<<divup(n,block_size*len),block_size>>>(d_sum,len,n);
      CHECK(hipGetLastError());
      std::cout<<"sum2"<<std::endl;
      CHECK(hipMemcpy(index,d_sum, n * sizeof(unsigned int), hipMemcpyDeviceToHost));
      for(int j=0; j<n; j++){
        std::cout<< index[j] << " " ;
      }
      std::cout<< std::endl;
      
      getIndex<<<divup(n,block_size*len),block_size>>>(d_index, d_sum, d_out, len, n, total_zeros);
      CHECK(hipGetLastError());
      std::cout<<"index"<<std::endl;
      CHECK(hipMemcpy(index,d_index, n * sizeof(unsigned int), hipMemcpyDeviceToHost));
      for(int j=0; j<n; j++){
        std::cout<< index[j] << " " ;
      }
      std::cout<< std::endl;

      scatter<<<divup(n,block_size*len),block_size>>>(d_in, d_index, d_out_long, len, n);
      // CHECK(cudaMemcpy(cal_result,d_out_long, n * sizeof(data_t), cudaMemcpyDeviceToHost));
      // for(int j=0; j<n; j++){
      //   std::cout<< cal_result[j] << " " ;
      // }
      // std::cout<< std::endl;
      CHECK(hipGetLastError());
      //must swap pointers
      d_temp = d_in;
      d_in = d_out_long;
      d_out_long = d_temp;
  }

  cuda_memcpy(data, d_in, n, hipMemcpyDeviceToHost);
  CHECK(hipFree(d_in));
  CHECK(hipFree(d_out_long));
  CHECK(hipFree(d_out));
  CHECK(hipFree(d_sum));
  CHECK(hipFree(d_index));
  // std::sort(data, data + n);
}